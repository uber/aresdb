//  Copyright (c) 2017-2018 Uber Technologies, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include <cstdio>
#include <cstring>
#include <exception>
#include <vector>
#include <initializer_list>
#include "query/transform.hpp"
#include "query/binder.hpp"

namespace ares {

// FilterContext is doing the actual filter after binding one or two
// input iterators.
template<typename FunctorType>
class FilterContext {
 public:
  FilterContext(
      uint8_t *predicateVector, int indexVectorLength,
      RecordID **foreignTableRecordIDVectors,
      int numForeignTables, FunctorType functorType,
      void *cudaStream)
      : predicateVector(predicateVector),
        indexVectorLength(indexVectorLength),
        foreignTableRecordIDVectors(foreignTableRecordIDVectors),
        numForeignTables(numForeignTables),
        functorType(functorType),
        cudaStream(reinterpret_cast<hipStream_t>(cudaStream)) {}

  hipStream_t getStream() const {
    return cudaStream;
  }

  template<typename InputIterator>
  int run(uint32_t *indexVector, InputIterator inputIterator);

  template<typename LHSIterator, typename RHSIterator>
  int run(uint32_t *indexVector, LHSIterator lhsIter, RHSIterator rhsIter);

 private:
  uint8_t *predicateVector;
  int indexVectorLength;
  RecordID **foreignTableRecordIDVectors;
  int numForeignTables;
  FunctorType functorType;
  hipStream_t cudaStream;

  template<typename LHSIterator, typename RHSIterator,
      typename IndexZipIterator>
  int executeRemoveIf(LHSIterator lhsIter,
                      RHSIterator rhsIter,
                      IndexZipIterator indexZipIterator);

  template<typename InputIterator, typename IndexZipIterator>
  int executeRemoveIf(InputIterator inputIter,
                      IndexZipIterator indexZipIterator);
};

}  // namespace ares

CGoCallResHandle UnaryFilter(InputVector input,
                             uint32_t *indexVector,
                             uint8_t *predicateVector,
                             int indexVectorLength,
                             RecordID **foreignTableRecordIDVectors,
                             int numForeignTables,
                             uint32_t *baseCounts,
                             uint32_t startCount,
                             UnaryFunctorType functorType,
                             void *cudaStream,
                             int device) {
  CGoCallResHandle resHandle = {nullptr, nullptr};
  try {
#ifdef RUN_ON_DEVICE
    hipSetDevice(device);
#endif
    ares::FilterContext<UnaryFunctorType> ctx(predicateVector,
                                              indexVectorLength,
                                              foreignTableRecordIDVectors,
                                              numForeignTables,
                                              functorType,
                                              cudaStream);
    std::vector<InputVector> inputVectors = {input};
    ares::InputVectorBinder<ares::FilterContext<UnaryFunctorType>, 1>
        binder(ctx, inputVectors, indexVector, baseCounts, startCount);
    resHandle.res =
        reinterpret_cast<void *>(binder.bind());
    CheckCUDAError("UnaryFilter");
  }
  catch (std::exception &e) {
    std::cerr << "Exception happend when doing UnaryFilter:" << e.what()
              << std::endl;
    resHandle.pStrErr = strdup(e.what());
  }
  return resHandle;
}

CGoCallResHandle BinaryFilter(InputVector lhs,
                              InputVector rhs,
                              uint32_t *indexVector,
                              uint8_t *predicateVector,
                              int indexVectorLength,
                              RecordID **foreignTableRecordIDVectors,
                              int numForeignTables,
                              uint32_t *baseCounts,
                              uint32_t startCount,
                              BinaryFunctorType functorType,
                              void *cudaStream,
                              int device) {
  CGoCallResHandle resHandle = {nullptr, nullptr};
  try {
#ifdef RUN_ON_DEVICE
    hipSetDevice(device);
#endif
    ares::FilterContext<BinaryFunctorType> ctx(predicateVector,
                                               indexVectorLength,
                                               foreignTableRecordIDVectors,
                                               numForeignTables,
                                               functorType,
                                               cudaStream);
    std::vector<InputVector> inputVectors = {lhs, rhs};
    ares::InputVectorBinder<ares::FilterContext<BinaryFunctorType>, 2> binder(
        ctx, inputVectors, indexVector, baseCounts, startCount);

    resHandle.res =
        reinterpret_cast<void *>(binder.bind());
    CheckCUDAError("BinaryFilter");
  }
  catch (std::exception &e) {
    std::cerr << "Exception happend when doing BinaryFilter:" << e.what()
              << std::endl;
    resHandle.pStrErr = strdup(e.what());
  }
  return resHandle;
}

namespace ares {

// Filter template function for unary transform filter.
template<typename FunctorType>
template<typename InputIterator, typename IndexZipIterator>
int FilterContext<FunctorType>::executeRemoveIf(
    InputIterator inputIter,
    IndexZipIterator indexZipIterator) {
  typedef typename InputIterator::value_type::head_type InputValueType;
  UnaryPredicateFunctor<bool, InputValueType> f(functorType);
  RemoveFilter<typename IndexZipIterator::value_type, uint8_t> removeFilter(
      predicateVector);
  // first compute the predicate values.
  thrust::transform(GET_EXECUTION_POLICY(cudaStream), inputIter,
                    inputIter + indexVectorLength, predicateVector, f);
  // then we use the predicate values to remove indexes in place.
  return thrust::remove_if(GET_EXECUTION_POLICY(cudaStream), indexZipIterator,
                           indexZipIterator + indexVectorLength, removeFilter) -
         indexZipIterator;
}

// run unary filter.
template<typename FunctorType>
template<typename InputIterator>
int FilterContext<FunctorType>::run(uint32_t *indexVector,
                                    InputIterator inputIterator) {
  switch (numForeignTables) {
    #define EXECUTE_UNARY_REMOVE_IF(NumTotalForeignTables) \
    case NumTotalForeignTables: { \
      IndexZipIteratorMaker<NumTotalForeignTables> maker; \
      return executeRemoveIf(inputIterator, \
                           maker.make(indexVector, \
                                      foreignTableRecordIDVectors)); \
    }

    EXECUTE_UNARY_REMOVE_IF(0)
    EXECUTE_UNARY_REMOVE_IF(1)
    EXECUTE_UNARY_REMOVE_IF(2)
    EXECUTE_UNARY_REMOVE_IF(3)
    EXECUTE_UNARY_REMOVE_IF(4)
    EXECUTE_UNARY_REMOVE_IF(5)
    EXECUTE_UNARY_REMOVE_IF(6)
    EXECUTE_UNARY_REMOVE_IF(7)
    EXECUTE_UNARY_REMOVE_IF(8)
    default:throw std::invalid_argument("only support up to 8 foreign tables");
  }
}

// run binary filter.
template<typename FunctorType>
template<typename LHSIterator, typename RHSIterator, typename IndexZipIterator>
int FilterContext<FunctorType>::executeRemoveIf(
    LHSIterator lhsIter,
    RHSIterator rhsIter,
    IndexZipIterator indexZipIterator) {

  typedef typename input_iterator_value_type<
        typename LHSIterator::value_type::head_type,
        typename RHSIterator::value_type::head_type>::type InputValueType1;
  typedef typename input_iterator_value_type<
        typename RHSIterator::value_type::head_type,
        typename LHSIterator::value_type::head_type>::type InputValueType2;

  BinaryPredicateFunctor<bool, InputValueType1, InputValueType2> f(functorType);
  RemoveFilter<typename IndexZipIterator::value_type, uint8_t> removeFilter(
      predicateVector);

  // first compute the predicate values.
  thrust::transform(GET_EXECUTION_POLICY(cudaStream), lhsIter,
      lhsIter + indexVectorLength, rhsIter, predicateVector, f);
  // then we use the predicate values to remove indexes in place.
  return thrust::remove_if(GET_EXECUTION_POLICY(cudaStream), indexZipIterator,
                           indexZipIterator + indexVectorLength, removeFilter) -
         indexZipIterator;
}

// template partial specialization with output iterator as uint8_t* for binary
// transform.
template<typename FunctorType>
template<typename LHSIterator, typename RHSIterator>
int FilterContext<FunctorType>::run(uint32_t *indexVector,
                                    LHSIterator lhsIter,
                                    RHSIterator rhsIter) {
  switch (numForeignTables) {
    #define EXECUTE_BINARY_REMOVE_IF(NumTotalForeignTables) \
    case NumTotalForeignTables: { \
      IndexZipIteratorMaker<NumTotalForeignTables> maker; \
      return executeRemoveIf(lhsIter, rhsIter, maker.make(indexVector, \
                               foreignTableRecordIDVectors)); \
    }

    EXECUTE_BINARY_REMOVE_IF(0)
    EXECUTE_BINARY_REMOVE_IF(1)
    EXECUTE_BINARY_REMOVE_IF(2)
    EXECUTE_BINARY_REMOVE_IF(3)
    EXECUTE_BINARY_REMOVE_IF(4)
    EXECUTE_BINARY_REMOVE_IF(5)
    EXECUTE_BINARY_REMOVE_IF(6)
    EXECUTE_BINARY_REMOVE_IF(7)
    EXECUTE_BINARY_REMOVE_IF(8)
    default:throw std::invalid_argument("only support up to 8 foreign tables");
  }
}

}  // namespace ares
