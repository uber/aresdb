//  Copyright (c) 2017-2018 Uber Technologies, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include <cstdio>
#include <cstring>
#include <exception>
#include <vector>
#include <initializer_list>
#include "query/transform.hpp"
#include "query/binder.hpp"

namespace ares {

// FilterContext is doing the actual filter after binding one or two
// input iterators.
template<typename FunctorType>
class FilterContext {
 public:
  FilterContext(
      uint8_t *predicateVector, int indexVectorLength,
      RecordID **foreignTableRecordIDVectors,
      int numForeignTables, FunctorType functorType,
      void *cudaStream)
      : predicateVector(predicateVector),
        indexVectorLength(indexVectorLength),
        foreignTableRecordIDVectors(foreignTableRecordIDVectors),
        numForeignTables(numForeignTables),
        functorType(functorType),
        cudaStream(reinterpret_cast<hipStream_t>(cudaStream)) {}

  hipStream_t getStream() const {
    return cudaStream;
  }

  template<typename InputIterator>
  int run(uint32_t *indexVector, InputIterator inputIterator) {
    switch (numForeignTables) {
      #define EXECUTE_UNARY_REMOVE_IF(NumTotalForeignTables) \
      case NumTotalForeignTables: { \
        IndexZipIteratorMaker<NumTotalForeignTables> maker; \
        return executeRemoveIf(inputIterator, \
                             maker.make(indexVector, \
                                        foreignTableRecordIDVectors)); \
      }

      EXECUTE_UNARY_REMOVE_IF(0)
      EXECUTE_UNARY_REMOVE_IF(1)
      EXECUTE_UNARY_REMOVE_IF(2)
      EXECUTE_UNARY_REMOVE_IF(3)
      EXECUTE_UNARY_REMOVE_IF(4)
      EXECUTE_UNARY_REMOVE_IF(5)
      EXECUTE_UNARY_REMOVE_IF(6)
      EXECUTE_UNARY_REMOVE_IF(7)
      EXECUTE_UNARY_REMOVE_IF(8)
      default:throw std::invalid_argument("only support up to 8 foreign tables");
    }
  }

  template<typename LHSIterator, typename RHSIterator>
  struct supported_combination {
  static constexpr bool value =
    ((std::is_same<typename LHSIterator::value_type::head_type, UUIDT*>::value &&
      (std::is_same<typename RHSIterator::value_type::head_type, UUIDT>::value ||
          std::is_same<typename RHSIterator::value_type::head_type, uint32_t>::value)) ||
    (std::is_same<typename LHSIterator::value_type::head_type, GeoPointT*>::value &&
      (std::is_same<typename RHSIterator::value_type::head_type, GeoPointT>::value ||
          std::is_same<typename RHSIterator::value_type::head_type, uint32_t>::value)) ||
    (!std::is_same<typename LHSIterator::value_type::head_type, UUIDT*>::value &&
      !std::is_same<typename LHSIterator::value_type::head_type, GeoPointT*>::value &&
      !std::is_same<typename RHSIterator::value_type::head_type, UUIDT*>::value &&
      !std::is_same<typename RHSIterator::value_type::head_type, GeoPointT*>::value));
  };

  template<typename LHSIterator, typename RHSIterator>
  typename std::enable_if<supported_combination<LHSIterator, RHSIterator>::value, int>::type
  run(uint32_t *indexVector, LHSIterator lhsIter, RHSIterator rhsIter) {
    switch (numForeignTables) {
      #define EXECUTE_BINARY_REMOVE_IF(NumTotalForeignTables) \
      case NumTotalForeignTables: { \
        IndexZipIteratorMaker<NumTotalForeignTables> maker; \
        return executeRemoveIf(lhsIter, rhsIter, maker.make(indexVector, \
                                foreignTableRecordIDVectors)); \
      }

      EXECUTE_BINARY_REMOVE_IF(0)
      EXECUTE_BINARY_REMOVE_IF(1)
      EXECUTE_BINARY_REMOVE_IF(2)
      EXECUTE_BINARY_REMOVE_IF(3)
      EXECUTE_BINARY_REMOVE_IF(4)
      EXECUTE_BINARY_REMOVE_IF(5)
      EXECUTE_BINARY_REMOVE_IF(6)
      EXECUTE_BINARY_REMOVE_IF(7)
      EXECUTE_BINARY_REMOVE_IF(8)
      default:throw std::invalid_argument("only support up to 8 foreign tables");
    }
  }

  template<typename LHSIterator, typename RHSIterator>
  typename std::enable_if<!supported_combination<LHSIterator, RHSIterator>::value, int>::type
  run(uint32_t *indexVector, LHSIterator lhsIter, RHSIterator rhsIter) {
    throw std::invalid_argument(
              "Unsupported data type combination" + std::to_string(__LINE__)
                  + "in filter context");
  }

 private:
  uint8_t *predicateVector;
  int indexVectorLength;
  RecordID **foreignTableRecordIDVectors;
  int numForeignTables;
  FunctorType functorType;
  hipStream_t cudaStream;

  template<typename LHSIterator, typename RHSIterator,
      typename IndexZipIterator>
  int executeRemoveIf(LHSIterator lhsIter,
                      RHSIterator rhsIter,
                      IndexZipIterator indexZipIterator);

  template<typename InputIterator, typename IndexZipIterator>
  int executeRemoveIf(InputIterator inputIter,
                      IndexZipIterator indexZipIterator);
};

}  // namespace ares

CGoCallResHandle UnaryFilter(InputVector input,
                             uint32_t *indexVector,
                             uint8_t *predicateVector,
                             int indexVectorLength,
                             RecordID **foreignTableRecordIDVectors,
                             int numForeignTables,
                             uint32_t *baseCounts,
                             uint32_t startCount,
                             UnaryFunctorType functorType,
                             void *cudaStream,
                             int device) {
  CGoCallResHandle resHandle = {nullptr, nullptr};
  try {
#ifdef RUN_ON_DEVICE
    hipSetDevice(device);
#endif
    ares::FilterContext<UnaryFunctorType> ctx(predicateVector,
                                              indexVectorLength,
                                              foreignTableRecordIDVectors,
                                              numForeignTables,
                                              functorType,
                                              cudaStream);
    std::vector<InputVector> inputVectors = {input};
    ares::InputVectorBinder<ares::FilterContext<UnaryFunctorType>, 1>
        binder(ctx, inputVectors, indexVector, baseCounts, startCount);
    resHandle.res =
        reinterpret_cast<void *>(binder.bind());
    CheckCUDAError("UnaryFilter");
  }
  catch (std::exception &e) {
    std::cerr << "Exception happend when doing UnaryFilter:" << e.what()
              << std::endl;
    resHandle.pStrErr = strdup(e.what());
  }
  return resHandle;
}

CGoCallResHandle BinaryFilter(InputVector lhs,
                              InputVector rhs,
                              uint32_t *indexVector,
                              uint8_t *predicateVector,
                              int indexVectorLength,
                              RecordID **foreignTableRecordIDVectors,
                              int numForeignTables,
                              uint32_t *baseCounts,
                              uint32_t startCount,
                              BinaryFunctorType functorType,
                              void *cudaStream,
                              int device) {
  CGoCallResHandle resHandle = {nullptr, nullptr};
  try {
#ifdef RUN_ON_DEVICE
    hipSetDevice(device);
#endif
    ares::FilterContext<BinaryFunctorType> ctx(predicateVector,
                                               indexVectorLength,
                                               foreignTableRecordIDVectors,
                                               numForeignTables,
                                               functorType,
                                               cudaStream);
    std::vector<InputVector> inputVectors = {lhs, rhs};
    ares::InputVectorBinder<ares::FilterContext<BinaryFunctorType>, 2> binder(
        ctx, inputVectors, indexVector, baseCounts, startCount);

    resHandle.res =
        reinterpret_cast<void *>(binder.bind());
    CheckCUDAError("BinaryFilter");
  }
  catch (std::exception &e) {
    std::cerr << "Exception happend when doing BinaryFilter:" << e.what()
              << std::endl;
    resHandle.pStrErr = strdup(e.what());
  }
  return resHandle;
}

namespace ares {

// Filter template function for unary transform filter.
template<typename FunctorType>
template<typename InputIterator, typename IndexZipIterator>
int FilterContext<FunctorType>::executeRemoveIf(
    InputIterator inputIter,
    IndexZipIterator indexZipIterator) {
  typedef typename InputIterator::value_type::head_type InputValueType;
  UnaryPredicateFunctor<bool, InputValueType> f(functorType);
  RemoveFilter<typename IndexZipIterator::value_type, uint8_t> removeFilter(
      predicateVector);
  // first compute the predicate values.
  thrust::transform(GET_EXECUTION_POLICY(cudaStream), inputIter,
                    inputIter + indexVectorLength, predicateVector, f);
  // then we use the predicate values to remove indexes in place.
  return thrust::remove_if(GET_EXECUTION_POLICY(cudaStream), indexZipIterator,
                           indexZipIterator + indexVectorLength, removeFilter) -
         indexZipIterator;
}

// run binary filter.
template<typename FunctorType>
template<typename LHSIterator, typename RHSIterator, typename IndexZipIterator>
int FilterContext<FunctorType>::executeRemoveIf(
    LHSIterator lhsIter,
    RHSIterator rhsIter,
    IndexZipIterator indexZipIterator) {

  typedef typename input_iterator_value_type<
        typename LHSIterator::value_type::head_type,
        typename RHSIterator::value_type::head_type>::type InputValueType1;
  typedef typename input_iterator_value_type<
        typename RHSIterator::value_type::head_type,
        typename LHSIterator::value_type::head_type>::type InputValueType2;

  BinaryPredicateFunctor<bool, InputValueType1, InputValueType2> f(functorType);
  RemoveFilter<typename IndexZipIterator::value_type, uint8_t> removeFilter(
      predicateVector);

  // first compute the predicate values.
  thrust::transform(GET_EXECUTION_POLICY(cudaStream), lhsIter,
      lhsIter + indexVectorLength, rhsIter, predicateVector, f);
  // then we use the predicate values to remove indexes in place.
  return thrust::remove_if(GET_EXECUTION_POLICY(cudaStream), indexZipIterator,
                           indexZipIterator + indexVectorLength, removeFilter) -
         indexZipIterator;
}
}  // namespace ares
